#include "hip/hip_runtime.h"
#include "algorithms_gpu.hpp"
#include <iostream>

__global__ void grayscale_kernel(unsigned char *data, int width, int height, int channels)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= width * height)
        return;

    unsigned char r = data[idx * channels];
    unsigned char g = data[idx * channels + 1];
    unsigned char b = data[idx * channels + 2];

    unsigned char gray = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);

    data[idx * channels] = gray;
    data[idx * channels + 1] = gray;
    data[idx * channels + 2] = gray;
}

__global__ void convolve_kernel(unsigned char *input, unsigned char *output, int width, int height, int channels, const float *kernel, int kernel_width, int kernel_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int kx_offset = kernel_width / 2;
    int ky_offset = kernel_height / 2;

    // Iterate over channels
    for (int c = 0; c < channels; c++)
    {
        float pixel_sum = 0.0f;

        // Apply kernel to the pixel and its neighbors
        for (int ky = 0; ky < kernel_height; ky++)
        {
            for (int kx = 0; kx < kernel_width; kx++)
            {
                int img_x = x + kx - kx_offset;
                int img_y = y + ky - ky_offset;

                // Ensure indices are within bounds
                if (img_x >= 0 && img_x < width && img_y >= 0 && img_y < height)
                {
                    int img_idx = (img_y * width + img_x) * channels + c;
                    int kernel_idx = ky * kernel_width + kx;
                    pixel_sum += input[img_idx] * kernel[kernel_idx];
                }
            }
        }

        // Clamp result to 0-255 and assign to output image
        int output_idx = (y * width + x) * channels + c;
        output[output_idx] = min(max(static_cast<int>(pixel_sum), 0), 255);
    }
}

__device__ void rgb_to_hsv_device(unsigned char r, unsigned char g, unsigned char b, float *h, float *s, float *v)
{
    float fr = r / 255.0f;
    float fg = g / 255.0f;
    float fb = b / 255.0f;

    float max_val = fmaxf(fmaxf(fr, fg), fb);
    float min_val = fminf(fminf(fr, fg), fb);
    float delta = max_val - min_val;

    // Calculate Value (V)
    *v = max_val;

    // Calculate Saturation (S)
    if (max_val == 0.0f)
    {
        *s = 0.0f;
    }
    else
    {
        *s = delta / max_val;
    }

    // Calculate Hue (H)
    if (delta == 0.0f)
    {
        *h = 0.0f; // Undefined hue
    }
    else if (max_val == fr)
    {
        *h = 60.0f * fmodf((fg - fb) / delta, 6.0f);
    }
    else if (max_val == fg)
    {
        *h = 60.0f * (((fb - fr) / delta) + 2.0f);
    }
    else
    {
        *h = 60.0f * (((fr - fg) / delta) + 4.0f);
    }

    if (*h < 0.0f)
    {
        *h += 360.0f;
    }
}

__global__ void rgb_to_hsv_kernel(unsigned char *input, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = (y * width + x) * channels;

    unsigned char r = input[idx];
    unsigned char g = input[idx + 1];
    unsigned char b = input[idx + 2];

    float h, s, v;
    rgb_to_hsv_device(r, g, b, &h, &s, &v);

    input[idx] = static_cast<unsigned char>(h / 360.0f * 255);
    input[idx + 1] = static_cast<unsigned char>(s * 255);
    input[idx + 2] = static_cast<unsigned char>(v * 255);
}

__device__ void rgb_to_ycbcr_device(unsigned char r, unsigned char g, unsigned char b, unsigned char *y, unsigned char *cb, unsigned char *cr)
{
    *y = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
    *cb = static_cast<unsigned char>(128 + (-0.168736f * r - 0.331264f * g + 0.5f * b));
    *cr = static_cast<unsigned char>(128 + (0.5f * r - 0.418688f * g - 0.081312f * b));
}

__global__ void rgb_to_ycbcr_kernel(unsigned char *input, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    int idx = (y * width + x) * channels;

    unsigned char r = input[idx];
    unsigned char g = input[idx + 1];
    unsigned char b = input[idx + 2];

    unsigned char y_val, cb_val, cr_val;
    rgb_to_ycbcr_device(r, g, b, &y_val, &cb_val, &cr_val);

    input[idx] = y_val;
    input[idx + 1] = cb_val;
    input[idx + 2] = cr_val;
}

void convert_to_grayscale_gpu(Image &image)
{
    // Check if image data is valid
    if (!image.data)
    {
        std::cerr << "Error: Image data pointer is null." << std::endl;
        exit(1);
    }

    // Check if dimensions are valid
    if (image.width <= 0 || image.height <= 0)
    {
        std::cerr << "Error: Invalid image dimensions. Width: " << image.width
                  << ", Height: " << image.height << std::endl;
        exit(1);
    }

    // Check if channels are valid
    if (image.channels < 3)
    {
        std::cerr << "Error: Image channels must be at least 3 (RGB). Channels: " << image.channels << std::endl;
        exit(1);
    }

    // Set up the grid and block dimensions
    int blockSize = 256;
    int gridSize = (image.width * image.height + blockSize - 1) / blockSize;

    if (gridSize <= 0)
    {
        std::cerr << "Error: Invalid grid size. Grid size: " << gridSize << std::endl;
        exit(1);
    }

    grayscale_kernel<<<gridSize, blockSize>>>(image.data, image.width, image.height, image.channels);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[grayscale_kernel] launch): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[grayscale_kernel] execution): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

void convolve_image_gpu(Image *image, const float *kernel, int kernel_width, int kernel_height)
{
    int img_size = image->width * image->height * image->channels;

    unsigned char *d_input = image->data;
    unsigned char *d_output;
    float *d_kernel;

    // Allocate memory for output and kernel on the GPU
    hipMalloc(&d_output, img_size * sizeof(unsigned char));
    hipMalloc(&d_kernel, kernel_width * kernel_height * sizeof(float));

    // Copy kernel data to the device
    hipMemcpy(d_kernel, kernel, kernel_width * kernel_height * sizeof(float), hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x, (image->height + blockDim.y - 1) / blockDim.y);

    convolve_kernel<<<gridDim, blockDim>>>(d_input, d_output, image->width, image->height, image->channels, d_kernel, kernel_width, kernel_height);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[convolve_kernel] launch): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[convolve_kernel] execution): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }

    // Free the original image data and update the image to point to the result in GPU memory
    hipFree(image->data);
    image->data = d_output;

    // Free the kernel memory
    hipFree(d_kernel);
}

void convert_image_rgb_to_hsv_gpu(Image *image)
{
    int img_size = image->width * image->height * image->channels;
    unsigned char *d_input = image->data;

    dim3 blockDim(16, 16);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x, (image->height + blockDim.y - 1) / blockDim.y);

    rgb_to_hsv_kernel<<<gridDim, blockDim>>>(d_input, image->width, image->height, image->channels);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[rgb_to_hsv_kernel] launch): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }

    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[rgb_to_hsv_kernel] execution): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

void convert_image_rgb_to_ycbcr_gpu(Image *image)
{
    int img_size = image->width * image->height * image->channels;
    unsigned char *d_input = image->data;

    // Set the grid and block dimensions
    dim3 blockDim(16, 16);
    dim3 gridDim((image->width + blockDim.x - 1) / blockDim.x, (image->height + blockDim.y - 1) / blockDim.y);

    // Launch the CUDA kernel
    rgb_to_ycbcr_kernel<<<gridDim, blockDim>>>(d_input, image->width, image->height, image->channels);

    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[rgb_to_ycbcr_kernel] launch): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }

    // Wait for the kernel to finish
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error (kernel[rgb_to_ycbcr_kernel] execution): " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}
