#include "common.hpp"
#include <iostream>

void transfer_to_gpu(Image *image)
{
    if (image->data == nullptr)
    {
        std::cerr << "Error: Image data is null, cannot transfer to GPU." << std::endl;
        exit(1);
    }

    // Allocate memory on the GPU
    unsigned char *d_data;
    size_t image_size = image->width * image->height * image->channels;
    hipError_t error = hipMalloc(&d_data, image_size);
    if (error != hipSuccess)
    {
        throw std::runtime_error("CUDA Error (hipMalloc): " + std::string(hipGetErrorString(error)));
    }

    // Copy the image data from CPU to GPU
    error = hipMemcpy(d_data, image->data, image_size, hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        throw std::runtime_error("CUDA Error (hipMemcpy HostToDevice): " + std::string(hipGetErrorString(error)));
    }

    // Free the CPU memory
    free_image_cpu(image);

    // Update the image structure to point to the GPU data
    image->data = d_data;
}

void transfer_to_cpu(Image *image)
{
    if (image->data == nullptr)
    {
        std::cerr << "Error: Image data is null, cannot transfer to CPU." << std::endl;
        exit(1);
    }

    // Allocate memory on the CPU
    unsigned char *h_data = new unsigned char[image->width * image->height * image->channels];

    // Copy the image data from GPU to CPU
    hipError_t error = hipMemcpy(h_data, image->data, image->width * image->height * image->channels, hipMemcpyDeviceToHost);
    if (error != hipSuccess)
    {
        throw std::runtime_error("CUDA Error (hipMemcpy DeviceToHost): " + std::string(hipGetErrorString(error)));
    }

    // Free the GPU memory
    error = hipFree(image->data);
    if (error != hipSuccess)
    {
        throw std::runtime_error("CUDA Error (hipFree): " + std::string(hipGetErrorString(error)));
    }

    // Update the image structure to point to the CPU data
    image->data = h_data;
}
