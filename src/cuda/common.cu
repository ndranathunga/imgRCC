#include "common.hpp"
#include <iostream>

void checkCudaError(hipError_t error, const char *message)
{
    if (error != hipSuccess)
    {
        std::cerr << "CUDA Error: " << message << " - " << hipGetErrorString(error) << std::endl;
        exit(1);
    }
}

void transfer_to_gpu(Image *image)
{
    if (image->data == nullptr)
    {
        std::cerr << "Error: Image data is null, cannot transfer to GPU." << std::endl;
        return;
    }

    // Allocate memory on the GPU
    unsigned char *d_data;
    size_t image_size = image->width * image->height * image->channels;
    hipError_t error = hipMalloc(&d_data, image_size);
    checkCudaError(error, "hipMalloc failed during transfer_to_gpu");

    // Copy the image data from CPU to GPU
    error = hipMemcpy(d_data, image->data, image_size, hipMemcpyHostToDevice);
    checkCudaError(error, "hipMemcpy HostToDevice failed during transfer_to_gpu");

    // Update the image structure to point to the GPU data
    image->data = d_data;
}

void transfer_to_cpu(Image *image)
{
    if (image->data == nullptr)
    {
        std::cerr << "Error: Image data is null, cannot transfer to CPU." << std::endl;
        return;
    }

    // Allocate memory on the CPU
    unsigned char *h_data = new unsigned char[image->width * image->height * image->channels];

    // Copy the image data from GPU to CPU
    hipError_t error = hipMemcpy(h_data, image->data, image->width * image->height * image->channels, hipMemcpyDeviceToHost);
    checkCudaError(error, "hipMemcpy DeviceToHost failed during transfer_to_cpu");

    // Free the GPU memory
    error = hipFree(image->data);
    checkCudaError(error, "hipFree failed during transfer_to_cpu");

    // Update the image structure to point to the CPU data
    image->data = h_data;
}
